#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../common/book.h"

#define N 10000

__global__ void add(int* a, int* b, int* c) {
	/*
	 *	blockIdx is the built-in variable which contains the value of
	 *	the block index for whichever block is currently running the
	 *	device code.
	 *	Another thing to be explained is the '.x' domain. In CUDA C,
	 *	blocks are actually defined in two-dimensions
	 */
	int tid = blockIdx.x;
	if(tid < N)	c[tid] = a[tid] + b[tid];
}

int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	for(int i = 0; i < N; ++i) {
		a[i] = -i;
		b[i] = i * i;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
 	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

	/*
	 *	Note the <<<N, 1>>> here:
	 *	The first N represents the number of parallel blocks in which we
	 *	we would like the device to execute out kernel. The runtime will
	 *	create N copies of the kernel and running them in parallel.
	 */
	add<<<N, 1>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	for(int i = 0; i < N; ++i) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
 	hipFree(dev_b);
 	hipFree(dev_c);

	return 0;
}
