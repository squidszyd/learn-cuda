#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../common/book.h"

__global__ void add(int a, int b, int* c) {
	*c = a + b;
}

int main(void) {
	int c;
	int* dev_c;
	/*
	 * ** DO NOT ** dereference the pointer returned by hipMalloc()
	 * from code that executes on the host !!
	 * RESTRICTIONS:
	 * <1>. Pass pointers allocated with hipMalloc() to functions that
	 *		execute on the device is allowed.
	 * <2>. It is allowed to read or write the pointers allocated with
	 *		hipMalloc() as long as they are run on the device.
	 * <3>. Pointers allocated with hipMalloc() can be passed to
	 *		functions execute on the host.
	 * <4>.	As the ** DO NOT ** says at line 12
	 */
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

	add<<<1, 1>>>(2, 7, dev_c);

	/*
	 *	hipMemcpyDeviceToHost		|
	 *	hipMemcpyHostToDevice		┠ As their names tell
	 *	hipMemcpyDeviceToDevice	|
	 */
	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

	printf("2 + 7 = %d\n", c);

	/*
	 * We must use hipFree to free the memory allocated by hipMalloc,
	 * but not C free.
	 */
	hipFree(dev_c);

	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	printf("Device count: %d\n", count);

	/*
	 *	hipDeviceProp_t is a structure contains information abount our device
	 *	char name[256], size_t totalGlobalMem and many more. [Page 28]
	 */
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	printf("Device name: %s\n", prop.name);

	return 0;
}
